#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <folly/init/Init.h>
#include <folly/logging/xlog.h>
#include <gtest/gtest.h>
#include "comms/common/IpcGpuBarrier.cuh"
#include "comms/common/algorithms/all_reduce/all_reduce_dda.cuh"
#include "comms/common/tests/TestBaselineBootstrap.h"
#include "comms/rcclx/develop/meta/lib/tests/RcclxTestUtils.h"
#include "comms/utils/CudaRAII.h"

using namespace meta::rcclx;
using namespace meta::comms;

namespace {
constexpr int NUMRANKS = 8;
const int cnt = 1024 * 1024;
const int nBlocks = 32;
const int nThreads = 128;
} // namespace

template <typename ElementType>
class AllReduceDdaTest : public RcclxBaseTestFixture {
 public:
  void SetUp() override {
    RcclxBaseTestFixture::SetUp();

    ncclConfig_t config = NCCL_CONFIG_INITIALIZER;
    NCCL_CHECK(
        ncclCommInitRankConfig(&comm, numRanks, commId, globalRank, &config));
    XLOGF(INFO, "rank {} init done; total ranks: {}", globalRank, numRanks);

    ASSERT_EQ(numRanks, NUMRANKS);
    CUDA_CHECK(hipSetDevice(globalRank));

    auto bootstrap = std::make_shared<TestBaselineBootstrap>(comm);
    memHandler =
        std::make_unique<IpcMemHandler>(bootstrap, globalRank, NUMRANKS);
    ipcBuf = std::make_unique<DeviceBuffer>(sizeof(ElementType) * cnt);
    memHandler->addSelfDeviceMemPtr(ipcBuf->get());
    memHandler->exchangeMemPtrs();

    void* ipcBufs[NUMRANKS];
    for (int i = 0; i < NUMRANKS; ++i) {
      ipcBufs[i] = memHandler->getPeerDeviceMemPtr(i);
    }
    allRankIpcBufs =
        std::make_unique<DeviceBuffer>(sizeof(ElementType*) * NUMRANKS);
    CUDA_CHECK(hipMemcpy(
        allRankIpcBufs->get(),
        ipcBufs,
        sizeof(ElementType*) * NUMRANKS,
        hipMemcpyHostToDevice));

    auto barrierInit =
        IpcGpuBarrier::mallocAndInit(numRanks, nBlocks, globalRank, bootstrap);
    barrierResources = std::move(barrierInit.first);
    barrier = std::move(barrierInit.second);
  }

  void TearDown() override {
    ncclCommFinalize(comm);
    ncclCommDestroy(comm);
    RcclxBaseTestFixture::TearDown();
  }

 public:
  ncclComm_t comm{nullptr};
  std::unique_ptr<DeviceBuffer> ipcBuf;
  std::unique_ptr<DeviceBuffer> allRankIpcBufs;
  std::unique_ptr<IpcMemHandler> memHandler;
  std::unique_ptr<IpcGpuBarrierResources> barrierResources;
  IpcGpuBarrier barrier;
};

TYPED_TEST_SUITE_P(AllReduceDdaTest);

// The range of the elements in the data arrary
const int RAND_RANGE = 100;

__global__ void initRand(hiprandState_t* randStates, int size, int nRanks) {
  for (auto idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    for (int i = 0; i < nRanks; i++) {
      // The init param of hiprandState is same for a given randState idx on any
      // ranks, so it will generate the same sequence of random numbers. This
      // allows us to compute the ground truth of allReduce locally without peer
      // rank communication.
      hiprand_init(
          i + 1 /* seed */,
          idx /* sequence */,
          0 /* offset */,
          &randStates[idx * nRanks + i]);
    }
  }
}

template <typename T>
__global__ void genData(
    hiprandState_t* randStates,
    T* data,
    T* acc,
    T* groundTruth,
    int selfRank,
    int nRanks,
    int size) {
  for (auto idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    // NOTE: the current DDA two-shot implementation assumes the bias vector is
    // same across all ranks.
    T bias = nRanks;
    acc[idx] = bias;
    T sum = bias;
    for (int i = 0; i < nRanks; ++i) {
      // populate data and then compute ground truth of the all reduce
      double val =
          hiprand_uniform_double(&randStates[idx * nRanks + i]) * RAND_RANGE;
      // downcast to T
      T hval = val;
      sum += hval;
      if (i == selfRank) {
        data[idx] = hval;
      }
    }
    groundTruth[idx] = sum;
  }
}

TYPED_TEST_P(AllReduceDdaTest, ddaAllReduceFlatIpcTest) {
  using ElementType = TypeParam;

  // The IpcGpuBarrier requires numThreads >= numRanks
  ASSERT_GE(nThreads, NUMRANKS);

  // we do 128 bit load in the dda kernel, so the data memory must align with
  // 128 bits
  ASSERT_EQ(cnt * sizeof(ElementType) % sizeof(uint4), 0);

  // prepare the sendbuff on each rank
  DeviceBuffer randStateBuf(sizeof(hiprandState_t) * NUMRANKS * cnt);
  hiprandState_t* randStates_d =
      reinterpret_cast<hiprandState_t*>(randStateBuf.get());
  initRand<<<nBlocks, nThreads>>>(randStates_d, cnt, NUMRANKS);

  DeviceBuffer sendbuf(sizeof(ElementType) * cnt);
  ElementType* sendbuf_d = reinterpret_cast<ElementType*>(sendbuf.get());

  DeviceBuffer accbuf(sizeof(ElementType) * cnt);
  ElementType* accbuf_d = reinterpret_cast<ElementType*>(accbuf.get());

  DeviceBuffer groundTruth(sizeof(ElementType) * cnt);
  ElementType* groundTruth_d =
      reinterpret_cast<ElementType*>(groundTruth.get());

  genData<<<nBlocks, nThreads>>>(
      randStates_d,
      sendbuf_d,
      accbuf_d,
      groundTruth_d,
      this->globalRank,
      NUMRANKS,
      cnt);

  DeviceBuffer recvbuff(sizeof(ElementType) * cnt);
  ElementType* recvbuff_d = reinterpret_cast<ElementType*>(recvbuff.get());
  ddaAllReduceFlatIpc<ElementType, NUMRANKS, true /*hasAcc*/>
      <<<nBlocks, nThreads>>>(
          (ElementType**)this->allRankIpcBufs->get(),
          recvbuff_d,
          cnt,
          sendbuf_d,
          this->globalRank,
          this->barrier,
          accbuf_d);

  hipDeviceSynchronize();

  // compare with ground truth
  ElementType myresults_h[cnt];
  ElementType groundTruth_h[cnt];
  CUDA_CHECK(hipMemcpy(
      myresults_h, recvbuff_d, sizeof(ElementType) * cnt, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(
      groundTruth_h,
      groundTruth_d,
      sizeof(ElementType) * cnt,
      hipMemcpyDefault));
  for (int i = 0; i < cnt; ++i) {
    EXPECT_EQ(
        static_cast<double>(myresults_h[i]),
        static_cast<double>(groundTruth_h[i]));
  }
}

TYPED_TEST_P(AllReduceDdaTest, ddaAllReduceTreeIpcTest) {
  using ElementType = TypeParam;

  // The IpcGpuBarrier requires numThreads >= numRanks
  ASSERT_GE(nThreads, NUMRANKS);
  // we do 128 bit load in the dda kernel and in two-shot algo each rank is
  // responsible for count/nRanks elements, so the data memory must align with
  // 128 bits
  ASSERT_EQ(cnt % NUMRANKS, 0);
  ASSERT_EQ(cnt / NUMRANKS * sizeof(ElementType) % sizeof(uint4), 0);

  // prepare the sendbuff on each rank
  DeviceBuffer randStateBuf(sizeof(hiprandState_t) * NUMRANKS * cnt);
  hiprandState_t* randStates_d =
      reinterpret_cast<hiprandState_t*>(randStateBuf.get());
  initRand<<<nBlocks, nThreads>>>(randStates_d, cnt, NUMRANKS);

  DeviceBuffer sendbuf(sizeof(ElementType) * cnt);
  ElementType* sendbuf_d = reinterpret_cast<ElementType*>(sendbuf.get());

  DeviceBuffer accbuf(sizeof(ElementType) * cnt);
  ElementType* accbuf_d = reinterpret_cast<ElementType*>(accbuf.get());

  DeviceBuffer groundTruth(sizeof(ElementType) * cnt);
  ElementType* groundTruth_d =
      reinterpret_cast<ElementType*>(groundTruth.get());

  genData<<<nBlocks, nThreads>>>(
      randStates_d,
      sendbuf_d,
      accbuf_d,
      groundTruth_d,
      this->globalRank,
      NUMRANKS,
      cnt);

  DeviceBuffer recvbuff(sizeof(ElementType) * cnt);
  ElementType* recvbuff_d = reinterpret_cast<ElementType*>(recvbuff.get());

  CUDA_CHECK(hipMemcpy(
      reinterpret_cast<ElementType**>(
          this->allRankIpcBufs->get())[this->globalRank],
      sendbuf_d,
      sizeof(ElementType) * cnt,
      hipMemcpyDefault));
  ddaAllReduceTreeIpc<ElementType, NUMRANKS, true /*hasAcc*/>
      <<<nBlocks, nThreads>>>(
          (ElementType**)this->allRankIpcBufs->get(),
          recvbuff_d,
          cnt,
          sendbuf_d,
          this->globalRank,
          this->barrier,
          accbuf_d);
  hipDeviceSynchronize();

  // compare with ground truth
  ElementType myresults_h[cnt];
  ElementType groundTruth_h[cnt];
  CUDA_CHECK(hipMemcpy(
      myresults_h, recvbuff_d, sizeof(ElementType) * cnt, hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy(
      groundTruth_h,
      groundTruth_d,
      sizeof(ElementType) * cnt,
      hipMemcpyDefault));
  for (int i = 0; i < cnt; ++i) {
    EXPECT_EQ(
        static_cast<double>(myresults_h[i]),
        static_cast<double>(groundTruth_h[i]));
  }
}

REGISTER_TYPED_TEST_SUITE_P(
    AllReduceDdaTest,
    ddaAllReduceFlatIpcTest,
    ddaAllReduceTreeIpcTest);
using TypesToTest = ::testing::Types<half, __hip_bfloat16>;
INSTANTIATE_TYPED_TEST_SUITE_P(
    AllReduceDdaTests,
    AllReduceDdaTest,
    TypesToTest);

int main(int argc, char* argv[]) {
  ::testing::InitGoogleTest(&argc, argv);
  ::testing::AddGlobalTestEnvironment(new MPIEnvironmentBase);
  folly::Init init(&argc, &argv);
  return RUN_ALL_TESTS();
}
