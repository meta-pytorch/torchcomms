#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda/atomic>
#include "comms/ctran/algos/common/GpeKernelSync.h"
#include "comms/ctran/algos/common/GpeKernelSyncDev.cuh"

using namespace ctran::algos;

__global__ void
waitSigTestKernel(GpeKernelSync* sync, uint64_t* data, int cmpVal) {
  const auto workerId = blockIdx.x;
  ::cuda::atomic_ref<uint64_t, cuda::thread_scope_system> ref{*data};
  if (threadIdx.x == 0) {
    while (ref.load(cuda::memory_order_acquire) != cmpVal)
      ;
    GpeKernelSyncDev::complete(sync, workerId, 0);
  }
}
