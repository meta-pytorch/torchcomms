#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "comms/ctran/algos/CtranAlgoDev.h"
#include "comms/ctran/algos/common/GpeKernelDev.cuh"

__global__ void ncclKernelGet(int* flag, CtranAlgoDeviceState* devState) {
  const auto gtIdx = blockDim.x * blockIdx.x + threadIdx.x;
  if (flag && gtIdx == 0) {
    ctran::device::KernelStartGpe(flag);
    ctran::device::KernelWaitGpeTerminate(flag);
  }
}
