// Copyright (c) Meta Platforms, Inc. and affiliates.
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include <folly/init/Init.h>

#include "hip/hip_runtime.h"
#include "mpi.h"
#include "nccl.h"

int main(int argc, char* argv[]) {
  folly::Init init(&argc, &argv);
  int size = 32 * 1024 * 1024;

  int localRank, globalRank, numRanks;

  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  std::tie(localRank, globalRank, numRanks, comm) = setupNccl(argc, argv);

  CUDACHECK_TEST(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK_TEST(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK_TEST(hipStreamCreate(&s));

  // communicating using NCCL
  NCCLCHECK_TEST(ncclAllGather(
      (const void*)sendbuff,
      (void*)recvbuff,
      size / numRanks,
      ncclFloat,
      comm,
      s));

  // completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK_TEST(hipStreamSynchronize(s));

  // free device buffers
  CUDACHECK_TEST(hipFree(sendbuff));
  CUDACHECK_TEST(hipFree(recvbuff));

  cleanupNccl(comm);

  printf("[MPI Rank %d] Success \n", globalRank);
  return 0;
}
